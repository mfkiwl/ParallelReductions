#include "hip/hip_runtime.h"
#include <vector>

#include <benchmark/benchmark.h>
#include <fmt/core.h>

#include "reduce_cpu.hpp"
#include "reduce_cuda.hpp"
#include "reduce_opencl.hpp"

using namespace av;
namespace bm = benchmark;
static std::vector<float> dataset;

template <typename accumulator_at> void generic(bm::State &state, accumulator_at&& accumulator) {
    double const sum_expected = dataset.size() * 1.0;
    double sum = 0;
    double error = 0;
    for (auto _ : state) {
        bm::DoNotOptimize(sum = accumulator());
        error = std::abs(sum_expected - sum)/sum_expected;
    }

    auto total_ops = state.iterations() * dataset.size();
    state.counters["adds/s"] = bm::Counter(total_ops, bm::Counter::kIsRate);
    state.counters["bytes/s"] = bm::Counter(total_ops * sizeof(float), bm::Counter::kIsRate);
    state.counters["error,%"] = bm::Counter(error * 100);
}

template <typename accumulator_at> 
void automatic(bm::State &state) {
    std::fill(dataset.begin(), dataset.end(), 1.f);
    accumulator_at acc {dataset.data(), dataset.data() + dataset.size()};
    generic(state, acc);
}

int main(int argc, char **argv) {

    // Parse configuration parameters.
    size_t elements = 0;
    if (argc <= 1) {
        fmt::print("You did not feed the size of arrays, so we will use a 1GB array!\n");
        elements = 1024 * 1024 * 1024 / sizeof(float);
    } else {
        elements = static_cast<size_t>(std::atol(argv[1]));
    }
    dataset.resize(elements);
    std::fill(dataset.begin(), dataset.end(), 1.f);

    // Register and run all the benchmarks.
    bm::RegisterBenchmark("cpu_baseline:f32", &automatic<cpu_baseline_gt<float>>)->MinTime(10);
    bm::RegisterBenchmark("cpu_baseline:f64", &automatic<cpu_baseline_gt<double>>)->MinTime(10);
    bm::RegisterBenchmark("cpu_avx2:f32", &automatic<cpu_avx2_f32_t>)->MinTime(10);
    bm::RegisterBenchmark("cpu_avx2:f32kahan", &automatic<cpu_avx2_kahan_t>)->MinTime(10);
    bm::RegisterBenchmark("cpu_avx2:f64", &automatic<cpu_avx2_f64_t>)->MinTime(10);
    bm::RegisterBenchmark("cpu_openmp", &automatic<cpu_openmp_t>)->MinTime(10);

    // Log available backends.
    auto ocl_targets = opencl_targets();
    for (auto const &tgt : ocl_targets)
        fmt::print("- OpenCL: {} ({}), {}, {}\n", tgt.device_name, tgt.device_version, tgt.driver_version,
                   tgt.language_version);

    if (cuda_device_count()) {
        bm::RegisterBenchmark("cuda_thrust", &automatic<cuda_thrust_t>)->MinTime(10);
        bm::RegisterBenchmark("cuda_tensors", &automatic<cuda_tensors_t>)->MinTime(10);
        bm::RegisterBenchmark("cuda_warps", &automatic<cuda_warps_t>)->MinTime(10);
        // bm::RegisterBenchmark("cuda_blocks", &automatic<cuda_blocks_t>)->MinTime(10);
    }
    else
        fmt::print("No CUDA capable devices found!\n");

    
    for (auto tgt : ocl_targets) {
        for (auto kernel_name : opencl_t::kernels_k) {
            for (auto group_size : opencl_wg_sizes) {
                auto name = fmt::format("opencl-{} split by {} on {}", kernel_name, group_size, tgt.device_name);
                bm::RegisterBenchmark(name.c_str(), [=](bm::State &state) {
                    opencl_t ocl(dataset.data(), dataset.data() + dataset.size(), tgt, group_size, kernel_name);
                    generic(state, ocl);
                })->MinTime(10);
            }
        }
    }


    bm::Initialize(&argc, argv);
    bm::RunSpecifiedBenchmarks();
    bm::Shutdown();
    return 0;
}